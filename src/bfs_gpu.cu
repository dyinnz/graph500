#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <set>
#include <string.h>
#include <queue>
//#include "translator_json_csr.h"
#include "wtime.h"
#include "graph.h"

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 32768
#define STAND_THREADS 256
#define STAND_BLOCKS 256
#define THREAD_BIN_SIZE 32

#define INF (int)(1<<30)

#define visited_color 1
#define unvisited_color 0
#define frontier_color 2

//#define first_alpha 4

typedef int bit_type;

const char output_file[] = "bfs_result.txt";
const char time_detail[] = "time_detail.csv";


int pivot_selection_first(
        graph *g,
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list,
        index_t *d_beg_pos,
        bit_type *d_vertex_status,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    //    index_t bin_offset = 0;
    //    index_t id = tid;
    //    index_t begin_offset = 0;
    //    if(id != 0)
    //        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_bw,
        index_t *d_adj_list_bw,
        index_t *d_beg_pos_bw,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    while(id < vertex_count)
    {
        if(d_vertex_status_bw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(d_vertex_status_bw[w] == level)
                {
                    d_vertex_status_bw[id] = level + 1;
                    *d_change = true;
                    break;
                    // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}



void bfs_cu(index_t root,
            index_t *beg_pos,
            const index_t vert_count,
            vertex_t *csr,
            const index_t edge_count)
{
    srand(time(NULL));
    hipSetDevice(0);

    graph *g = new graph(beg_pos, vert_count, csr, edge_count);
    const index_t vertex_count = g->vert_count + 1;

    bit_type *backward_vertex_status = (bit_type *)malloc(sizeof(bit_type)*vertex_count);
    index_t *frontier_queue = (index_t*)malloc(sizeof(index_t)*vertex_count);

    //------------------------------------------------------------------------
    //Deciding how many blocks to be used
    index_t number_of_blocks = 1;
    index_t number_of_threads_per_block = vertex_count;

    if(vertex_count > THREADS_PER_BLOCK)
    {
        number_of_blocks = (index_t)ceil(vertex_count/(double)THREADS_PER_BLOCK);
        number_of_threads_per_block = THREADS_PER_BLOCK;
        if(number_of_blocks > BLOCKS_PER_GRID)
            number_of_blocks = BLOCKS_PER_GRID;
    }

    //    printf("blocks = %d, threads = %d\n", number_of_blocks, number_of_threads_per_block);
    //------------------------------------------------------------------------
    //allocating auxiliars in CPU
    for(index_t i = 0 ; i < vertex_count ; ++i)
    {
        backward_vertex_status[i] = 0;//no colors
    }
    backward_vertex_status[root] = 1;
    //------------------------------------------------------------------------
    //Allocating GPU memory:
    printf("root = %d\n", root);
    index_t *d_adj_list_reverse;
    hipMalloc((void**) &d_adj_list_reverse, sizeof(index_t)*edge_count);
    hipMemcpy( d_adj_list_reverse, g->csr, sizeof(index_t)*edge_count, hipMemcpyHostToDevice);

    index_t *d_beg_pos_reverse;
    hipMalloc((void**) &d_beg_pos_reverse, sizeof(index_t)*(vertex_count + 1));
    hipMemcpy( d_beg_pos_reverse, g->beg_pos, sizeof(index_t)*(vertex_count + 1), hipMemcpyHostToDevice);

    bit_type *d_backward_vertex_status;
    hipMalloc((void**) &d_backward_vertex_status, sizeof(bit_type)*vertex_count);
    hipMemcpy( d_backward_vertex_status, backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyHostToDevice);

    index_t *d_vertex_count;
    hipMalloc((void **) &d_vertex_count, sizeof(index_t));
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);

    index_t offset = 0;
    index_t *d_offset;
    hipMalloc((void **) &d_offset, sizeof(index_t));
    hipMemcpy(d_offset, &(offset), sizeof(index_t), hipMemcpyHostToDevice);

    index_t *d_frontier_queue;
    hipMalloc((void **) &d_frontier_queue, sizeof(index_t) * vertex_count);

    index_t *d_thread_bin;
    hipMalloc((void **) &d_thread_bin, sizeof(index_t) * STAND_BLOCKS * STAND_THREADS * THREAD_BIN_SIZE);
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    //CPU & GPU shared variable

    index_t * level;
    index_t * d_level;
    hipHostAlloc((void **) &level, sizeof(index_t), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_level, level, 0);
    *level = 0;

    bool * change;
    bool * d_change;
    hipHostAlloc((void **) &change, sizeof(bool), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_change, change, 0);
    *change = true;

    FILE * fp_time = fopen(time_detail, "w");
    double time = wtime();

    double bfs_bw_time;
    do {
        while(*level < first_alpha)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_sync_color_top_down_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_backward_vertex_status,
                    d_change,
                    d_vertex_count,
                    d_level,
                    d_frontier_queue,
                    d_thread_bin);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);

            if (!*change) {
               break;
            }

        }

        bfs_bw_time = wtime() - time;
        printf("depth = %d\n", *level);
        printf("bfs top down time = %g (ms)\n", bfs_bw_time * 1000);

        if (!*change) {
           break;
        }


        while(*change)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_bottom_up_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_backward_vertex_status,
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_change,
                    d_vertex_count,
                    d_level);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);
        }
    } while(*change);

    bfs_bw_time = wtime() - time;
    fclose(fp_time);
    printf("depth = %d\n", *level);
    printf("bfs bottom up time = %g (ms)\n", bfs_bw_time * 1000);
    printf("teps %g (ms)\n", edge_count / bfs_bw_time * 1000);
    hipMemcpy(backward_vertex_status, d_backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyDeviceToHost);
    FILE * fp_out = fopen(output_file, "w");
    for(index_t i=0; i<vertex_count; ++i)
    {
        fprintf(fp_out, "%d %d\n", i, backward_vertex_status[i]);
    }
    fclose(fp_out);

    free(backward_vertex_status);
    hipFree(d_adj_list_reverse);
    hipFree(d_beg_pos_reverse);
    hipFree(d_backward_vertex_status);
    hipFree(d_change);
}


