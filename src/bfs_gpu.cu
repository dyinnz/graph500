#include "hip/hip_runtime.h"
/*
int pivot_selection_first(
        graph *g,
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list,
        index_t *d_beg_pos,
        bit_type *d_vertex_status,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    //    index_t bin_offset = 0;
    //    index_t id = tid;
    //    index_t begin_offset = 0;
    //    if(id != 0)
    //        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_bw,
        index_t *d_adj_list_bw,
        index_t *d_beg_pos_bw,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    while(id < vertex_count)
    {
        if(d_vertex_status_bw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(d_vertex_status_bw[w] == level)
                {
                    d_vertex_status_bw[id] = level + 1;
                    *d_change = true;
                    break;
                    // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}
*/

/*----------------------------------------------------------------------------*/
// distributed graph for cuda

#include "bfs_gpu.h"

#include <sys/time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

inline double wtime()
{
	double time[2];
	struct timeval time1;
	gettimeofday(&time1, NULL);

	time[0]=time1.tv_sec;
	time[1]=time1.tv_usec;

	return time[0]+time[1]*1.0e-6;
}

// global device variables

__device__ int      gd_mpi_rank;
__device__ int      gd_mpi_size;
__device__ int64_t  gd_local_v_num;
__device__ int64_t  gd_global_v_num;
__device__ int64_t  gd_local_v_beg;
__device__ int64_t  gd_local_v_end;
__device__ int64_t  gd_average;


/*----------------------------------------------------------------------------*/
// initializtion and finalizaiton


static void InitHostInfo(HostInfo &host_info) {
  // buffer size of all gather
  host_info.average = host_info.global_v_num / settings.mpi_size;
  host_info.change = 0;

  memset(host_info.bfs_tree, -1, sizeof(int64_t) * host_info.local_v_num);
}


static void InitCudaDevice(CudaInfo &cuda_info) {
  // TODO: change this value
  cuda_info.blocks_number = 8;
  cuda_info.threads_per_block = 64;

  hipSetDevice(settings.mpi_rank % 4);
}


static void HostAllocMemory(HostInfo &host_info) {
  host_info.local_bitmap = new bit_type[host_info.local_v_num];
  host_info.global_bitmap = new bit_type[host_info.global_v_num];

  memset(host_info.local_bitmap, 0, sizeof(bit_type) * host_info.local_v_num);
  memset(host_info.global_bitmap, 0, sizeof(bit_type) * host_info.global_v_num);
}


static void HostFreeMemory(HostInfo &host_info) {
  delete []host_info.local_bitmap;
  delete []host_info.global_bitmap;
}


static void CudaAllocMemory(HostInfo &host_info, CudaGraphMemory &d_graph) {

  // alloc and copy
  hipMalloc((void**)&d_graph.adja_arrays,
      sizeof(int64_t) * host_info.local_v_num * 2);
  hipMalloc((void**)&d_graph.csr, sizeof(int64_t) * host_info.csr_edge_num);

  hipMemcpy(d_graph.adja_arrays, host_info.adja_arrays,
      sizeof(int64_t) * host_info.local_v_num * 2, hipMemcpyHostToDevice);
  hipMemcpy(d_graph.csr, host_info.csr,
      sizeof(int64_t) * host_info.csr_edge_num, hipMemcpyHostToDevice);

  // alloc and clear
  hipMalloc((void**)&d_graph.bfs_tree,
      sizeof(int64_t) * host_info.local_v_num);
  hipMalloc((void**)&d_graph.local_bitmap,
      sizeof(bit_type) * host_info.local_v_num);
  hipMalloc((void**)&d_graph.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num);
  hipMalloc((void**)&d_graph.p_change, sizeof(bool));

  hipMemset(d_graph.bfs_tree, -1, sizeof(int64_t) * host_info.local_v_num);
  hipMemset(d_graph.local_bitmap, 0,
      sizeof(bit_type) * host_info.local_v_num);
  // hipMemset(d_graph.global_bitmap, 0,
      // sizeof(bit_type) * host_info.global_v_num);
  hipMemset(d_graph.p_change, 0, sizeof(bool));

  // just copy
  hipMemcpyToSymbol(HIP_SYMBOL(gd_mpi_rank), &settings.mpi_rank, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_mpi_size), &settings.mpi_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_num), &host_info.local_v_num, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_global_v_num), &host_info.global_v_num, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_beg), &host_info.local_v_beg, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_end), &host_info.local_v_end, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_average), &host_info.average, sizeof(int64_t));
}


static void CudaFreeMemory(CudaGraphMemory &d_graph) {
  hipFree(d_graph.adja_arrays);
  hipFree(d_graph.csr);
  hipFree(d_graph.bfs_tree);
  hipFree(d_graph.local_bitmap);
  hipFree(d_graph.global_bitmap);
  hipFree(d_graph.p_change);
}


/*----------------------------------------------------------------------------*/


static void MPIGatherAllBitmap(HostInfo &host_info, CudaGraphMemory &d_graph) {

  MPI_Allgather(host_info.local_bitmap, host_info.average, MPI_INT,
      host_info.global_bitmap, host_info.average, MPI_INT,
      MPI_COMM_WORLD);

  // the last process send the remainder vertexes to others
  int64_t remainder = host_info.global_v_num % settings.mpi_size;
  if (0 != remainder) {

    if (settings.mpi_rank == settings.mpi_size-1) {
      bit_type *send_buff = host_info.local_bitmap + host_info.local_v_num
        - remainder;
      MPI_Bcast(send_buff, remainder, MPI_INT, settings.mpi_size-1,
          MPI_COMM_WORLD);

    } else {
      bit_type *recv_buff = host_info.global_bitmap + host_info.global_v_num
        - remainder;
      MPI_Bcast(recv_buff, remainder, MPI_INT, settings.mpi_size-1,
          MPI_COMM_WORLD);
    }
  }
}


static void SetBFSRoot(HostInfo &host_info, CudaGraphMemory &d_graph) {

  for (int v = 0; v < host_info.global_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }

  if (host_info.local_v_beg <= host_info.root && host_info.root < host_info.local_v_end) {
    // set root

    int64_t local_root = host_info.root - host_info.local_v_beg;
    host_info.bfs_tree[local_root] = host_info.root;
    host_info.local_bitmap[local_root] = true;

    hipMemcpy(d_graph.bfs_tree, host_info.bfs_tree,
        sizeof(int64_t) * host_info.local_v_num, hipMemcpyHostToDevice);
    hipMemcpy(d_graph.local_bitmap, host_info.local_bitmap,
        sizeof(bit_type) * host_info.local_v_num, hipMemcpyHostToDevice);
  }

  // sync with others

  MPIGatherAllBitmap(host_info, d_graph);

  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyHostToDevice);

  for (int v = 0; v < host_info.global_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }
}


static void SyncWithMPI(HostInfo &host_info, CudaGraphMemory &d_graph) {

  // without gpu direct

  /*------ bitmap ------*/
  hipMemcpy(host_info.local_bitmap, d_graph.local_bitmap,
      sizeof(bit_type) * host_info.local_v_num, hipMemcpyDeviceToHost);

  for (int v = 0; v < host_info.local_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's local bitmap %ld\n", v + host_info.local_v_beg,
        host_info.local_bitmap[v]);
  }

  MPIGatherAllBitmap(host_info, d_graph);

  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyHostToDevice);

  for (int v = 0; v < host_info.global_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }


  /*------ change ------*/
  hipMemcpy(&host_info.change, d_graph.p_change,
      sizeof(bool), hipMemcpyDeviceToHost);
  MPI_Allreduce(MPI_IN_PLACE, &host_info.change, 1, MPI_BYTE,
      MPI_BOR, MPI_COMM_WORLD);
  // hipMemcpy(d_graph.p_change, &host_info.change,
      // sizeof(bool), hipMemcpyHostToDevice);
}



static void CopyBFSTree(HostInfo &host_info, CudaGraphMemory &d_graph) {
  hipMemcpy(host_info.bfs_tree, d_graph.bfs_tree,
      sizeof(int64_t) * host_info.local_v_num, hipMemcpyDeviceToHost);

  for (int64_t v = 0; v < host_info.local_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's parent %ld\n", v + host_info.local_v_beg,
        host_info.bfs_tree[v]);
  }
}


static __device__ int64_t local_to_global(int64_t local) {
  return gd_local_v_beg + local;
}


static __device__ int64_t global_to_local(int64_t global) {
  return global - gd_local_v_beg;
}


static __global__ void BFSTopDown() {
  // TODO:
}


static __global__ void BFSBottomUp(
    int64_t *adja_arrays,
    int64_t *csr,
    int64_t *bfs_tree,
    bit_type *local_bitmap,
    bit_type *global_bitmap,
    bool *p_change) {

  const int64_t local_v_num  = gd_local_v_num;
  const int64_t kThreadsNumber = blockDim.x * gridDim.x;

  *p_change = false;

  for (int64_t local_v = blockIdx.x*blockDim.x + threadIdx.x;
      local_v < local_v_num; local_v += kThreadsNumber) {

    // unvisited
    int64_t global_v = local_to_global(local_v);
    if (-1 == bfs_tree[local_v]) {

      /*printf("v [%ld] not visited, beg %ld, end %ld\n", */
          /*global_v, adja_arrays[2*local_v], adja_arrays[2*local_v+1]);*/

      for (int64_t iter = adja_arrays[2*local_v];
          iter < adja_arrays[2*local_v+1]; ++iter) {

        int64_t global_u = csr[iter];
        /*printf("v[%ld] <-> u[%ld]\n", global_v, global_u);*/

        // its parent havs been visited
        if (global_bitmap[global_u]) {

          /*printf("get v[%ld] 's parent global_u %ld\n", global_v, global_u);*/

          local_bitmap[local_v] = true;
          bfs_tree[local_v] = global_u;

          *p_change = true;
          break;
        }
      }
    }
  }
}


void CudaBFS(int64_t root,
             int64_t *adja_arrays,
             int64_t local_v_num,
             int64_t global_v_num,
             int64_t local_v_beg,
             int64_t local_v_end,
             int64_t *csr,
             int64_t csr_edge_num,
             int64_t *bfs_tree) {

  HostInfo host_info {
    root,
    adja_arrays,
    local_v_num,
    global_v_num,
    local_v_beg,
    local_v_end,
    csr,
    csr_edge_num,
    bfs_tree,
    0,
    0,
    0,
    0,
  };
  CudaInfo cuda_info;
  CudaGraphMemory d_graph;

  InitHostInfo(host_info);
  InitCudaDevice(cuda_info);

  HostAllocMemory(host_info);
  CudaAllocMemory(host_info, d_graph);

  /*--------------------------------------------------------------------------*/
  // debug print
  logger.mpi_debug("root:         %ld\n", host_info.root);
  logger.mpi_debug("local_v_num:  %ld\n", host_info.local_v_num);
  logger.mpi_debug("global_v_num: %ld\n", host_info.global_v_num);
  logger.mpi_debug("local_v_beg:  %ld\n", host_info.local_v_beg);
  logger.mpi_debug("local_v_end:  %ld\n", host_info.local_v_end);
  logger.mpi_debug("csr_edge_num: %ld\n", host_info.csr_edge_num);
  logger.mpi_debug("average:      %ld\n", host_info.average);
  logger.mpi_debug("remainder:    %ld\n",
      host_info.global_v_num % settings.mpi_size);
  /*--------------------------------------------------------------------------*/

  SetBFSRoot(host_info, d_graph);

  for (int64_t v = 0; v < host_info.local_v_num; ++v) {
    logger.mpi_debug("v[%ld] 's parent %ld; global bitmap %d\n",
        v + host_info.local_v_beg, host_info.bfs_tree[v],
        host_info.global_bitmap[v+host_info.local_v_beg]);
  }


  double time = wtime();
  do {

    if (false) {
      BFSTopDown<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          );
    } else {
      BFSBottomUp<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          d_graph.adja_arrays,
          d_graph.csr,
          d_graph.bfs_tree,
          d_graph.local_bitmap,
          d_graph.global_bitmap,
          d_graph.p_change);
    }

    SyncWithMPI(host_info, d_graph);

  } while (host_info.change);

  double bfs_bw_time = wtime() - time;
  logger.log("bfs time %lf\n", bfs_bw_time*1000);

  CopyBFSTree(host_info, d_graph);

  hipDeviceSynchronize();

  CudaFreeMemory(d_graph);
  HostFreeMemory(host_info);
}

