#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <set>
#include <string.h>
#include <queue>
//#include "translator_json_csr.h"
#include "wtime.h"
#include "graph.h"

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 32768
#define STAND_THREADS 256
#define STAND_BLOCKS 256
#define THREAD_BIN_SIZE 32

#define INF (int)(1<<30)

#define visited_color 1
#define unvisited_color 0
#define frontier_color 2

//#define first_alpha 4

typedef int bit_type;

const char output_file[] = "bfs_result.txt";
const char time_detail[] = "time_detail.csv";


int pivot_selection_first(
        graph *g,
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list,
        index_t *d_beg_pos,
        bit_type *d_vertex_status,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    //    index_t bin_offset = 0;
    //    index_t id = tid;
    //    index_t begin_offset = 0;
    //    if(id != 0)
    //        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_bw,
        index_t *d_adj_list_bw,
        index_t *d_beg_pos_bw,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    while(id < vertex_count)
    {
        if(d_vertex_status_bw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(d_vertex_status_bw[w] == level)
                {
                    d_vertex_status_bw[id] = level + 1;
                    *d_change = true;
                    break;
                    // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}



void bfs_cu(index_t root,
            index_t *beg_pos,
            const index_t vert_count,
            vertex_t *csr,
            const index_t edge_count,
            const index_t first_alpha=2)
{
    srand(time(NULL));
    hipSetDevice(0);

    graph *g = new graph(beg_pos, vert_count, csr, edge_count);
    const index_t vertex_count = g->vert_count + 1;

    bit_type *backward_vertex_status = (bit_type *)malloc(sizeof(bit_type)*vertex_count);
    index_t *frontier_queue = (index_t*)malloc(sizeof(index_t)*vertex_count);

    //------------------------------------------------------------------------
    //Deciding how many blocks to be used
    index_t number_of_blocks = 1;
    index_t number_of_threads_per_block = vertex_count;

    if(vertex_count > THREADS_PER_BLOCK)
    {
        number_of_blocks = (index_t)ceil(vertex_count/(double)THREADS_PER_BLOCK);
        number_of_threads_per_block = THREADS_PER_BLOCK;
        if(number_of_blocks > BLOCKS_PER_GRID)
            number_of_blocks = BLOCKS_PER_GRID;
    }

    //    printf("blocks = %d, threads = %d\n", number_of_blocks, number_of_threads_per_block);
    //------------------------------------------------------------------------
    //allocating auxiliars in CPU
    for(index_t i = 0 ; i < vertex_count ; ++i)
    {
        backward_vertex_status[i] = 0;//no colors
    }
    backward_vertex_status[root] = 1;
    //------------------------------------------------------------------------
    //Allocating GPU memory:
    printf("root = %ld\n", root);
    index_t *d_adj_list_reverse;
    hipMalloc((void**) &d_adj_list_reverse, sizeof(index_t)*edge_count);
    hipMemcpy( d_adj_list_reverse, g->csr, sizeof(index_t)*edge_count, hipMemcpyHostToDevice);

    index_t *d_beg_pos_reverse;
    hipMalloc((void**) &d_beg_pos_reverse, sizeof(index_t)*(vertex_count + 1));
    hipMemcpy( d_beg_pos_reverse, g->beg_pos, sizeof(index_t)*(vertex_count + 1), hipMemcpyHostToDevice);

    bit_type *d_backward_vertex_status;
    hipMalloc((void**) &d_backward_vertex_status, sizeof(bit_type)*vertex_count);
    hipMemcpy( d_backward_vertex_status, backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyHostToDevice);

    index_t *d_vertex_count;
    hipMalloc((void **) &d_vertex_count, sizeof(index_t));
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);

    index_t offset = 0;
    index_t *d_offset;
    hipMalloc((void **) &d_offset, sizeof(index_t));
    hipMemcpy(d_offset, &(offset), sizeof(index_t), hipMemcpyHostToDevice);

    index_t *d_frontier_queue;
    hipMalloc((void **) &d_frontier_queue, sizeof(index_t) * vertex_count);

    index_t *d_thread_bin;
    hipMalloc((void **) &d_thread_bin, sizeof(index_t) * STAND_BLOCKS * STAND_THREADS * THREAD_BIN_SIZE);
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    //CPU & GPU shared variable

    index_t * level;
    index_t * d_level;
    hipHostAlloc((void **) &level, sizeof(index_t), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_level, level, 0);
    *level = 0;

    bool * change;
    bool * d_change;
    hipHostAlloc((void **) &change, sizeof(bool), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_change, change, 0);
    *change = true;

    FILE * fp_time = fopen(time_detail, "w");
    double time = wtime();

    double bfs_bw_time;
    do {
        while(*level < first_alpha)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_sync_color_top_down_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_backward_vertex_status,
                    d_change,
                    d_vertex_count,
                    d_level,
                    d_frontier_queue,
                    d_thread_bin);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);

            if (!*change) {
               break;
            }

        }

        bfs_bw_time = wtime() - time;
        printf("depth = %d\n", *level);
        printf("bfs top down time = %g (ms)\n", bfs_bw_time * 1000);

        if (!*change) {
           break;
        }


        while(*change)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_bottom_up_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_backward_vertex_status,
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_change,
                    d_vertex_count,
                    d_level);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);
        }
    } while(*change);

    bfs_bw_time = wtime() - time;
    fclose(fp_time);
    printf("depth = %d\n", *level);
    printf("bfs bottom up time = %g (ms)\n", bfs_bw_time * 1000);
    printf("teps %g (ms)\n", edge_count / bfs_bw_time * 1000);
    hipMemcpy(backward_vertex_status, d_backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyDeviceToHost);
    FILE * fp_out = fopen(output_file, "w");
    for(index_t i=0; i<vertex_count; ++i)
    {
        fprintf(fp_out, "%d %d\n", i, backward_vertex_status[i]);
    }
    fclose(fp_out);

    free(backward_vertex_status);
    hipFree(d_adj_list_reverse);
    hipFree(d_beg_pos_reverse);
    hipFree(d_backward_vertex_status);
    hipFree(d_change);
}


struct CudaInfo {
};

struct HostInformation {
  int64_t root;
  int64_t *adja_arrays;
  int64_t local_v_num;
  int64_t global_v_num;
  int64_t *csr;
  int64_t csr_edge_num;
  int64_t *bfs_tree;
};

struct CudaGraphMemory {
  int64_t *adja_arrays;
  int64_t *local_v_num;
  int64_t *global_v_num;
  int64_t *csr;
  int64_t *bfs_tree;
};

/*----------------------------------------------------------------------------*/

void InitCudaDevice(CudaInfo &cuda_info) {
}

void HostAllocMemory(HostInformation &host_info) {
}

void HostFreeMemory(HostInformation &host_info) {
}

void SyncWithMPI(HostInformation &host_info, CudaGraphMemory &d_graph) {

}

void CudaAllocMemory(HostInformation &host_info, CudaGraphMemory &d_graph) {
  hipMalloc((void**)d_graph.adja_arrays, 
      sizeof(int64_t) * host_info.local_v_num);
  hipMalloc((void**)d_graph.local_v_num, sizeof(int64_t));
  hipMalloc((void**)d_graph.global_v_num, sizeof(int64_t));
  hipMalloc((void**)d_graph.csr, sizeof(int64_t) * host_info.csr_edge_num);
  hipMalloc((void**)d_graph.bfs_tree, 
      sizeof(int64_t) * host_info.local_v_num);

  hipMemcpy(d_graph.adja_arrays, host_info.adja_arrays,
      sizeof(int64_t) * host_info.local_v_num, hipMemcpyHostToDevice);
  hipMemcpy(d_graph.local_v_num, &host_info.local_v_num, 
      sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_graph.global_v_num, &host_info.global_v_num, 
      sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_graph.csr, host_info.csr, 
      sizeof(int64_t) * host_info.csr_edge_num, hipMemcpyHostToDevice);
}

void CudaFreeMemory(CudaGraphMemory &d_graph) {
  hipFree(d_graph.adja_arrays);
  hipFree(d_graph.local_v_num);
  hipFree(d_graph.global_v_num);
  hipFree(d_graph.csr);
  hipFree(d_graph.bfs_tree);
}

__global__ void BFSTopDown() {

}

__global__ void BFSBottomUp() {

}

void CudaBFS(int64_t root, 
             int64_t *adja_arrays, 
             int64_t local_v_num, 
             int64_t global_v_num,
             int64_t *csr,
             int64_t csr_edge_num,
             int64_t *bfs_tree) {
  CudaInfo cuda_info;
  HostInformation host_info = {
      root,
      adja_arrays,
      local_v_num,
      global_v_num,
      csr,
      csr_edge_num,
      bfs_tree,
  };
  CudaGraphMemory d_graph;

  InitCudaDevice(cuda_info);

  HostAllocMemory(host_info);
  CudaAllocMemory(host_info, d_graph);

  do {

    if (false) {
      BFSTopDown<<<1,1>>>();
    } else {
      BFSBottomUp<<<1,1>>>();
    }

    SyncWithMPI(host_info, d_graph);

  } while (false);
 
  CudaFreeMemory(d_graph);
  HostFreeMemory(host_info);
}


