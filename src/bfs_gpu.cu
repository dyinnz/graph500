#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <set>
#include <string.h>
#include <queue>
#include <mpi.h>
//#include "translator_json_csr.h"
#include "wtime.h"
#include "graph.h"

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 32768
#define STAND_THREADS 256
#define STAND_BLOCKS 256
#define THREAD_BIN_SIZE 32

#define INF (int)(1<<30)

#define visited_color 1
#define unvisited_color 0
#define frontier_color 2

//#define first_alpha 4

typedef int bit_type;

const char output_file[] = "bfs_result.txt";
const char time_detail[] = "time_detail.csv";


int pivot_selection_first(
        graph *g,
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list,
        index_t *d_beg_pos,
        bit_type *d_vertex_status,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    //    index_t bin_offset = 0;
    //    index_t id = tid;
    //    index_t begin_offset = 0;
    //    if(id != 0)
    //        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_bw,
        index_t *d_adj_list_bw,
        index_t *d_beg_pos_bw,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    while(id < vertex_count)
    {
        if(d_vertex_status_bw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(d_vertex_status_bw[w] == level)
                {
                    d_vertex_status_bw[id] = level + 1;
                    *d_change = true;
                    break;
                    // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}



void bfs_cu(index_t root,
            index_t *beg_pos,
            const index_t vert_count,
            vertex_t *csr,
            const index_t edge_count,
            const index_t first_alpha=2)
{
    srand(time(NULL));
    hipSetDevice(0);

    graph *g = new graph(beg_pos, vert_count, csr, edge_count);
    const index_t vertex_count = g->vert_count + 1;

    bit_type *backward_vertex_status = (bit_type *)malloc(sizeof(bit_type)*vertex_count);
    index_t *frontier_queue = (index_t*)malloc(sizeof(index_t)*vertex_count);

    //------------------------------------------------------------------------
    //Deciding how many blocks to be used
    index_t number_of_blocks = 1;
    index_t number_of_threads_per_block = vertex_count;

    if(vertex_count > THREADS_PER_BLOCK)
    {
        number_of_blocks = (index_t)ceil(vertex_count/(double)THREADS_PER_BLOCK);
        number_of_threads_per_block = THREADS_PER_BLOCK;
        if(number_of_blocks > BLOCKS_PER_GRID)
            number_of_blocks = BLOCKS_PER_GRID;
    }

    //    printf("blocks = %d, threads = %d\n", number_of_blocks, number_of_threads_per_block);
    //------------------------------------------------------------------------
    //allocating auxiliars in CPU
    for(index_t i = 0 ; i < vertex_count ; ++i)
    {
        backward_vertex_status[i] = 0;//no colors
    }
    backward_vertex_status[root] = 1;
    //------------------------------------------------------------------------
    //Allocating GPU memory:
    printf("root = %ld\n", root);
    index_t *d_adj_list_reverse;
    hipMalloc((void**) &d_adj_list_reverse, sizeof(index_t)*edge_count);
    hipMemcpy( d_adj_list_reverse, g->csr, sizeof(index_t)*edge_count, hipMemcpyHostToDevice);

    index_t *d_beg_pos_reverse;
    hipMalloc((void**) &d_beg_pos_reverse, sizeof(index_t)*(vertex_count + 1));
    hipMemcpy( d_beg_pos_reverse, g->beg_pos, sizeof(index_t)*(vertex_count + 1), hipMemcpyHostToDevice);

    bit_type *d_backward_vertex_status;
    hipMalloc((void**) &d_backward_vertex_status, sizeof(bit_type)*vertex_count);
    hipMemcpy( d_backward_vertex_status, backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyHostToDevice);

    index_t *d_vertex_count;
    hipMalloc((void **) &d_vertex_count, sizeof(index_t));
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);

    index_t offset = 0;
    index_t *d_offset;
    hipMalloc((void **) &d_offset, sizeof(index_t));
    hipMemcpy(d_offset, &(offset), sizeof(index_t), hipMemcpyHostToDevice);

    index_t *d_frontier_queue;
    hipMalloc((void **) &d_frontier_queue, sizeof(index_t) * vertex_count);

    index_t *d_thread_bin;
    hipMalloc((void **) &d_thread_bin, sizeof(index_t) * STAND_BLOCKS * STAND_THREADS * THREAD_BIN_SIZE);
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    //CPU & GPU shared variable

    index_t * level;
    index_t * d_level;
    hipHostAlloc((void **) &level, sizeof(index_t), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_level, level, 0);
    *level = 0;

    bool * change;
    bool * d_change;
    hipHostAlloc((void **) &change, sizeof(bool), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_change, change, 0);
    *change = true;

    FILE * fp_time = fopen(time_detail, "w");
    double time = wtime();

    double bfs_bw_time;
    do {
        while(*level < first_alpha)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_sync_color_top_down_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_backward_vertex_status,
                    d_change,
                    d_vertex_count,
                    d_level,
                    d_frontier_queue,
                    d_thread_bin);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);

            if (!*change) {
               break;
            }

        }

        bfs_bw_time = wtime() - time;
        printf("depth = %d\n", *level);
        printf("bfs top down time = %g (ms)\n", bfs_bw_time * 1000);

        if (!*change) {
           break;
        }


        while(*change)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_bottom_up_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_backward_vertex_status,
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_change,
                    d_vertex_count,
                    d_level);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);
        }
    } while(*change);

    bfs_bw_time = wtime() - time;
    fclose(fp_time);
    printf("depth = %d\n", *level);
    printf("bfs bottom up time = %g (ms)\n", bfs_bw_time * 1000);
    printf("teps %g (ms)\n", edge_count / bfs_bw_time * 1000);
    hipMemcpy(backward_vertex_status, d_backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyDeviceToHost);
    FILE * fp_out = fopen(output_file, "w");
    for(index_t i=0; i<vertex_count; ++i)
    {
        fprintf(fp_out, "%d %d\n", i, backward_vertex_status[i]);
    }
    fclose(fp_out);

    free(backward_vertex_status);
    hipFree(d_adj_list_reverse);
    hipFree(d_beg_pos_reverse);
    hipFree(d_backward_vertex_status);
    hipFree(d_change);
}

/*----------------------------------------------------------------------------*/
// distributed graph for cuda

// global device variables

__device__ int      gd_mpi_rank;
__device__ int      gd_mpi_size;
__device__ int64_t  gd_local_v_num;
__device__ int64_t  gd_global_v_num;
__device__ int64_t  gd_local_v_beg;
__device__ int64_t  gd_local_v_end;
__device__ int64_t  gd_average;

struct CudaInfo {
  int64_t blocks_number;
  int64_t threads_per_block;
};

struct HostInfo {
  int64_t mpi_rank;
  int64_t mpi_size;
  int64_t root;
  int64_t *adja_arrays;
  int64_t local_v_num;
  int64_t global_v_num;
  int64_t local_v_beg;
  int64_t local_v_end;
  int64_t *csr;
  int64_t csr_edge_num;
  int64_t *bfs_tree;
  // calc
  int64_t average;
  bool change;
  /*int64_t mpi_local_v_num;*/
  /*int64_t mpi_global_v_num;*/
  // tmp
  bit_type *local_bitmap;
  bit_type *global_bitmap;
};

struct CudaGraphMemory {
  int64_t *adja_arrays;
  int64_t *csr;
  int64_t *bfs_tree;
  bit_type *local_bitmap;
  bit_type *global_bitmap;
  bool *p_change;
};

/*----------------------------------------------------------------------------*/

void InitHostInfo(HostInfo &host_info) {
  // buffer size of all gather
  /*
  host_info.mpi_local_v_num = host_info.local_v_num;
  MPI_Allreduce(MPI_IN_PLACE, &host_info.local_v_num, 1, MPI_LONG_LONG,
      MPI_MAX, MPI_COMM_WORLD);
  host_info.mpi_global_v_num = host_info.mpi_local_v_num * host_info.mpi_size;
  */
  host_info.average = host_info.global_v_num / host_info.mpi_size;
  host_info.change = 0;
}

void InitCudaDevice(CudaInfo &cuda_info) {
  // TODO: change this value
  cuda_info.blocks_number = 256; 
  cuda_info.threads_per_block = 256;
}

void HostAllocMemory(HostInfo &host_info) {
  host_info.local_bitmap = new bit_type[host_info.local_v_num];
  host_info.global_bitmap = new bit_type[host_info.global_v_num];
}

void HostFreeMemory(HostInfo &host_info) {
  delete []host_info.local_bitmap;
  delete []host_info.local_bitmap;
}

void SyncWithMPI(HostInfo &host_info, CudaGraphMemory &d_graph) {


  // without gpu direct
  // bottom up
  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyDeviceToHost);

  // all mpi processes have the average vertexes at least
  MPI_Allgather(host_info.local_bitmap, host_info.average, MPI_BYTE,
      host_info.global_bitmap, host_info.average, MPI_BYTE,
      MPI_COMM_WORLD);
  
  int64_t remainder = host_info.global_v_num % host_info.mpi_size;
  bit_type *address = host_info.global_bitmap + host_info.global_v_num 
    - remainder;
  MPI_Scatter(address, remainder, MPI_BYTE, address, remainder, MPI_BYTE,
      host_info.mpi_size-1, MPI_COMM_WORLD);

  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap, 
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyHostToDevice);
}

void CudaAllocMemory(HostInfo &host_info, CudaGraphMemory &d_graph) {
  // alloc and copy
  hipMalloc((void**)&d_graph.adja_arrays, 
      sizeof(int64_t) * host_info.local_v_num);
  hipMalloc((void**)&d_graph.csr, sizeof(int64_t) * host_info.csr_edge_num);

  hipMemcpy(d_graph.adja_arrays, host_info.adja_arrays,
      sizeof(int64_t) * host_info.local_v_num, hipMemcpyHostToDevice);
  hipMemcpy(d_graph.csr, host_info.csr, 
      sizeof(int64_t) * host_info.csr_edge_num, hipMemcpyHostToDevice);

  // alloc and clear
  hipMalloc((void**)&d_graph.bfs_tree,
      sizeof(int64_t) * host_info.local_v_num);
  hipMalloc((void**)&d_graph.local_bitmap,
      sizeof(bit_type) * host_info.local_v_num);
      /*sizeof(bit_type) * host_info.mpi_local_v_num);*/
  hipMalloc((void**)&d_graph.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num);
      /*sizeof(bit_type) * host_info.mpi_global_v_num);*/
  hipMalloc((void**)&d_graph.p_change, sizeof(bool));
  hipMemset(d_graph.bfs_tree, -1, sizeof(int64_t) * host_info.local_v_num);
  hipMemset(d_graph.local_bitmap, 0, 
      sizeof(bit_type) * host_info.local_v_num);
      /*sizeof(bit_type) * host_info.mpi_local_v_num);*/
  hipMemset(d_graph.global_bitmap, 0,
      sizeof(bit_type) * host_info.global_v_num);
      /*sizeof(bit_type) * host_info.mpi_global_v_num);*/
  hipMemset(d_graph.p_change, 0, sizeof(bool));

  // just copy
  hipMemcpy(&gd_mpi_rank, &host_info.mpi_rank, 
      sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&gd_mpi_size, &host_info.mpi_size, 
      sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&gd_local_v_num, &host_info.local_v_num, 
      sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(&gd_global_v_num, &host_info.global_v_num, 
      sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(&gd_local_v_beg, &host_info.local_v_beg, 
      sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(&gd_local_v_end, &host_info.local_v_end, 
      sizeof(int64_t), hipMemcpyHostToDevice);

  int64_t average = host_info.global_v_num / host_info.mpi_size;
  hipMemcpy(&gd_average, &average, 
      sizeof(int64_t), hipMemcpyHostToDevice);
}

void CudaFreeMemory(CudaGraphMemory &d_graph) {
  hipFree(d_graph.adja_arrays);
  hipFree(d_graph.csr);
  hipFree(d_graph.bfs_tree);
  hipFree(d_graph.local_bitmap);
  hipFree(d_graph.global_bitmap);
}

__device__ int64_t local_to_global(int64_t local) {
  return gd_local_v_beg + local;
}

__device__ int64_t global_to_local(int64_t global) {
  return global - gd_local_v_beg;
}

__global__ void BFSTopDown() {

}

__global__ void BFSBottomUp(int64_t *adja_arrays,
                            int64_t *csr,
                            int64_t *bfs_tree,
                            bit_type *local_bitmap,
                            bit_type *global_bitmap) {

  const int64_t local_v_num  = gd_local_v_num;
  // const int64_t global_v_num = *p_global_v_num;
  const int64_t kThreadsNumber = blockDim.x * gridDim.x;

  for (int64_t local_u = blockIdx.x*blockDim.x + threadIdx.x;
      local_u < local_v_num; local_u += kThreadsNumber) {

    // unvisited
    int64_t global_u = local_to_global(local_u);
    if (!global_bitmap[global_u]) {

      for (int64_t offset = adja_arrays[global_u]; 
          offset < adja_arrays[global_u+1]; ++offset) {

        // its parent havs been visited
        int64_t global_v = csr[offset];
        if (-1 != bfs_tree[global_v]) {

          int64_t local_v = global_to_local(global_v);
          local_bitmap[local_v] = true;
          bfs_tree[local_v] = global_u;
        }
      }
    }
  }
}

void CudaBFS(int mpi_rank,
             int mpi_size,
             int64_t root, 
             int64_t *adja_arrays, 
             int64_t local_v_num, 
             int64_t global_v_num,
             int64_t local_v_beg,
             int64_t local_v_end,
             int64_t *csr,
             int64_t csr_edge_num,
             int64_t *bfs_tree) {
  CudaInfo cuda_info;
  HostInfo host_info = {
    mpi_rank,
    mpi_size,
    root,
    adja_arrays,
    local_v_num,
    global_v_num,
    local_v_beg,
    local_v_end,
    csr,
    csr_edge_num,
    bfs_tree,
    0,
    0,
    0,
    0,
  };
  CudaGraphMemory d_graph;

  InitHostInfo(host_info);
  InitCudaDevice(cuda_info);

  HostAllocMemory(host_info);
  CudaAllocMemory(host_info, d_graph);

  do {

    if (false) {
      BFSTopDown<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          );
    } else {
      BFSBottomUp<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          d_graph.adja_arrays,
          d_graph.csr,
          d_graph.bfs_tree,
          d_graph.local_bitmap,
          d_graph.global_bitmap
          );
    }

    SyncWithMPI(host_info, d_graph);

  } while (false);
 
  CudaFreeMemory(d_graph);
  HostFreeMemory(host_info);
}


