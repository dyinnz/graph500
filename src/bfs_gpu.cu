#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <set>
#include <string.h>
#include <queue>
#include <mpi.h>
//#include "translator_json_csr.h"
#include "wtime.h"
#include "graph.h"

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 32768
#define STAND_THREADS 256
#define STAND_BLOCKS 256
#define THREAD_BIN_SIZE 32

#define INF (int)(1<<30)

#define visited_color 1
#define unvisited_color 0
#define frontier_color 2

//#define first_alpha 4

typedef int bit_type;

const char output_file[] = "bfs_result.txt";
const char time_detail[] = "time_detail.csv";


int pivot_selection_first(
        graph *g,
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list,
        index_t *d_beg_pos,
        bit_type *d_vertex_status,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    //    index_t bin_offset = 0;
    //    index_t id = tid;
    //    index_t begin_offset = 0;
    //    if(id != 0)
    //        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_bw,
        index_t *d_adj_list_bw,
        index_t *d_beg_pos_bw,
        bool *d_change,
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
    while(id < vertex_count)
    {
        if(d_vertex_status_bw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(d_vertex_status_bw[w] == level)
                {
                    d_vertex_status_bw[id] = level + 1;
                    *d_change = true;
                    break;
                    // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}



void bfs_cu(index_t root,
            index_t *beg_pos,
            const index_t vert_count,
            vertex_t *csr,
            const index_t edge_count,
            const index_t first_alpha=2)
{
    srand(time(NULL));
    hipSetDevice(0);

    graph *g = new graph(beg_pos, vert_count, csr, edge_count);
    const index_t vertex_count = g->vert_count + 1;

    bit_type *backward_vertex_status = (bit_type *)malloc(sizeof(bit_type)*vertex_count);
    index_t *frontier_queue = (index_t*)malloc(sizeof(index_t)*vertex_count);

    //------------------------------------------------------------------------
    //Deciding how many blocks to be used
    index_t number_of_blocks = 1;
    index_t number_of_threads_per_block = vertex_count;

    if(vertex_count > THREADS_PER_BLOCK)
    {
        number_of_blocks = (index_t)ceil(vertex_count/(double)THREADS_PER_BLOCK);
        number_of_threads_per_block = THREADS_PER_BLOCK;
        if(number_of_blocks > BLOCKS_PER_GRID)
            number_of_blocks = BLOCKS_PER_GRID;
    }

    //    printf("blocks = %d, threads = %d\n", number_of_blocks, number_of_threads_per_block);
    //------------------------------------------------------------------------
    //allocating auxiliars in CPU
    for(index_t i = 0 ; i < vertex_count ; ++i)
    {
        backward_vertex_status[i] = 0;//no colors
    }
    backward_vertex_status[root] = 1;
    //------------------------------------------------------------------------
    //Allocating GPU memory:
    printf("root = %ld\n", root);
    index_t *d_adj_list_reverse;
    hipMalloc((void**) &d_adj_list_reverse, sizeof(index_t)*edge_count);
    hipMemcpy( d_adj_list_reverse, g->csr, sizeof(index_t)*edge_count, hipMemcpyHostToDevice);

    index_t *d_beg_pos_reverse;
    hipMalloc((void**) &d_beg_pos_reverse, sizeof(index_t)*(vertex_count + 1));
    hipMemcpy( d_beg_pos_reverse, g->beg_pos, sizeof(index_t)*(vertex_count + 1), hipMemcpyHostToDevice);

    bit_type *d_backward_vertex_status;
    hipMalloc((void**) &d_backward_vertex_status, sizeof(bit_type)*vertex_count);
    hipMemcpy( d_backward_vertex_status, backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyHostToDevice);

    index_t *d_vertex_count;
    hipMalloc((void **) &d_vertex_count, sizeof(index_t));
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);

    index_t offset = 0;
    index_t *d_offset;
    hipMalloc((void **) &d_offset, sizeof(index_t));
    hipMemcpy(d_offset, &(offset), sizeof(index_t), hipMemcpyHostToDevice);

    index_t *d_frontier_queue;
    hipMalloc((void **) &d_frontier_queue, sizeof(index_t) * vertex_count);

    index_t *d_thread_bin;
    hipMalloc((void **) &d_thread_bin, sizeof(index_t) * STAND_BLOCKS * STAND_THREADS * THREAD_BIN_SIZE);
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    //CPU & GPU shared variable

    index_t * level;
    index_t * d_level;
    hipHostAlloc((void **) &level, sizeof(index_t), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_level, level, 0);
    *level = 0;

    bool * change;
    bool * d_change;
    hipHostAlloc((void **) &change, sizeof(bool), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_change, change, 0);
    *change = true;

    FILE * fp_time = fopen(time_detail, "w");
    double time = wtime();

    double bfs_bw_time;
    do {
        while(*level < first_alpha)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_sync_color_top_down_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_backward_vertex_status,
                    d_change,
                    d_vertex_count,
                    d_level,
                    d_frontier_queue,
                    d_thread_bin);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);

            if (!*change) {
               break;
            }

        }

        bfs_bw_time = wtime() - time;
        printf("depth = %d\n", *level);
        printf("bfs top down time = %g (ms)\n", bfs_bw_time * 1000);

        if (!*change) {
           break;
        }


        while(*change)
        {
            (*level) ++;
            *change = false;
            double temp_time_beg = wtime();

            bfs_bottom_up_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                    d_backward_vertex_status,
                    d_adj_list_reverse,
                    d_beg_pos_reverse,
                    d_change,
                    d_vertex_count,
                    d_level);

            hipDeviceSynchronize();
            double temp_time = wtime() - temp_time_beg;
            fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);
        }
    } while(*change);

    bfs_bw_time = wtime() - time;
    fclose(fp_time);
    printf("depth = %d\n", *level);
    printf("bfs bottom up time = %g (ms)\n", bfs_bw_time * 1000);
    printf("teps %g (ms)\n", edge_count / bfs_bw_time * 1000);
    hipMemcpy(backward_vertex_status, d_backward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyDeviceToHost);
    FILE * fp_out = fopen(output_file, "w");
    for(index_t i=0; i<vertex_count; ++i)
    {
        fprintf(fp_out, "%d %d\n", i, backward_vertex_status[i]);
    }
    fclose(fp_out);

    free(backward_vertex_status);
    hipFree(d_adj_list_reverse);
    hipFree(d_beg_pos_reverse);
    hipFree(d_backward_vertex_status);
    hipFree(d_change);
}

/*----------------------------------------------------------------------------*/
// distributed graph for cuda

// global device variables

__device__ int      gd_mpi_rank;
__device__ int      gd_mpi_size;
__device__ int64_t  gd_local_v_num;
__device__ int64_t  gd_global_v_num;
__device__ int64_t  gd_local_v_beg;
__device__ int64_t  gd_local_v_end;
__device__ int64_t  gd_average;

struct CudaInfo {
  int64_t blocks_number;
  int64_t threads_per_block;
};

struct HostInfo {
  int64_t mpi_rank;
  int64_t mpi_size;
  int64_t root;
  int64_t *adja_arrays;
  int64_t local_v_num;
  int64_t global_v_num;
  int64_t local_v_beg;
  int64_t local_v_end;
  int64_t *csr;
  int64_t csr_edge_num;
  int64_t *bfs_tree;
  // calc
  int64_t average;
  bool change;
  /*int64_t mpi_local_v_num;*/
  /*int64_t mpi_global_v_num;*/
  // tmp
  bit_type *local_bitmap;
  bit_type *global_bitmap;
};

struct CudaGraphMemory {
  int64_t *adja_arrays;
  int64_t *csr;
  int64_t *bfs_tree;
  bit_type *local_bitmap;
  bit_type *global_bitmap;
  bool *p_change;
};

#define debug_print(__format, ...) do { \
  printf("RANK-%d %s(): "__format, host_info.mpi_rank, __func__, ##__VA_ARGS__); \
} while (false);

/*----------------------------------------------------------------------------*/

void InitHostInfo(HostInfo &host_info) {
  // buffer size of all gather
  host_info.average = host_info.global_v_num / host_info.mpi_size;
  host_info.change = 0;

  memset(host_info.bfs_tree, -1, sizeof(int64_t) * host_info.local_v_num);
}

void InitCudaDevice(CudaInfo &cuda_info) {
  // TODO: change this value
  cuda_info.blocks_number = 8;
  cuda_info.threads_per_block = 64;
}

void HostAllocMemory(HostInfo &host_info) {
  host_info.local_bitmap = new bit_type[host_info.local_v_num];
  host_info.global_bitmap = new bit_type[host_info.global_v_num];

  memset(host_info.local_bitmap, 0, sizeof(bit_type) * host_info.local_v_num);
  memset(host_info.global_bitmap, 0, sizeof(bit_type) * host_info.global_v_num);
}

void HostFreeMemory(HostInfo &host_info) {
  delete []host_info.local_bitmap;
  delete []host_info.global_bitmap;
}

void SyncWithMPI(HostInfo &host_info, CudaGraphMemory &d_graph) {

  // without gpu direct

  /*------ bitmap ------*/
  hipMemcpy(host_info.local_bitmap, d_graph.local_bitmap,
      sizeof(bit_type) * host_info.local_v_num, hipMemcpyDeviceToHost);

  for (int v = 0; v < host_info.local_v_num; ++v) {
    debug_print("v[%ld] 's local bitmap %ld\n", v + host_info.local_v_beg,
        host_info.local_bitmap[v]);
  }

  // all mpi processes have the average vertexes at least
  MPI_Allgather(host_info.local_bitmap, host_info.average, MPI_INT,
      host_info.global_bitmap, host_info.average, MPI_INT,
      MPI_COMM_WORLD);

  // the last process send the remainder vertexes to others
  int64_t remainder = host_info.global_v_num % host_info.mpi_size;
  if (0 != remainder) {

    if (host_info.mpi_rank == host_info.mpi_size-1) {
      bit_type *send_buff = host_info.local_bitmap + host_info.local_v_num
        - remainder;
      MPI_Bcast(send_buff, remainder, MPI_INT, host_info.mpi_size-1, 
          MPI_COMM_WORLD);

    } else {
      bit_type *recv_buff = host_info.global_bitmap + host_info.global_v_num 
        - remainder;
      MPI_Bcast(recv_buff, remainder, MPI_INT, host_info.mpi_size-1,
          MPI_COMM_WORLD);
    }
  }

  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap, 
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyHostToDevice);


  /*------ change ------*/
  hipMemcpy(&host_info.change, d_graph.p_change,
      sizeof(bool), hipMemcpyDeviceToHost);
  MPI_Allreduce(MPI_IN_PLACE, &host_info.change, 1, MPI_BYTE, 
      MPI_BOR, MPI_COMM_WORLD);
  hipMemcpy(d_graph.p_change, &host_info.change, 
      sizeof(bool), hipMemcpyHostToDevice);

  for (int v = 0; v < host_info.global_v_num; ++v) {
    debug_print("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }
}

void SetBFSRoot(HostInfo &host_info, CudaGraphMemory &d_graph) {
  for (int v = 0; v < host_info.global_v_num; ++v) {
    debug_print("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }

  if (host_info.local_v_beg <= host_info.root && host_info.root < host_info.local_v_end) {
    int64_t local_root = host_info.root - host_info.local_v_beg;
    host_info.bfs_tree[local_root] = host_info.root;
    host_info.local_bitmap[local_root] = true;

    hipMemcpy(d_graph.bfs_tree, host_info.bfs_tree, 
        sizeof(int64_t) * host_info.local_v_num, hipMemcpyHostToDevice);
    hipMemcpy(d_graph.local_bitmap, host_info.local_bitmap, 
        sizeof(bit_type) * host_info.local_v_num, hipMemcpyHostToDevice);
  }

  MPI_Allgather(host_info.local_bitmap, host_info.average, MPI_INT,
      host_info.global_bitmap, host_info.average, MPI_INT,
      MPI_COMM_WORLD);

  // the last process send the remainder vertexes to others
  int64_t remainder = host_info.global_v_num % host_info.mpi_size;
  if (0 != remainder) {

    if (host_info.mpi_rank == host_info.mpi_size-1) {
      bit_type *send_buff = host_info.local_bitmap + host_info.local_v_num
        - remainder;
      MPI_Bcast(send_buff, remainder, MPI_INT, host_info.mpi_size-1, 
          MPI_COMM_WORLD);

    } else {
      bit_type *recv_buff = host_info.global_bitmap + host_info.global_v_num 
        - remainder;
      MPI_Bcast(recv_buff, remainder, MPI_INT, host_info.mpi_size-1,
          MPI_COMM_WORLD);
    }
  }

  hipMemcpy(d_graph.global_bitmap, host_info.global_bitmap, 
      sizeof(bit_type) * host_info.global_v_num, hipMemcpyHostToDevice);

  for (int v = 0; v < host_info.global_v_num; ++v) {
    debug_print("v[%ld] 's global bitmap %ld\n", v,
        host_info.global_bitmap[v]);
  }
}

void CopyBFSTree(HostInfo &host_info, CudaGraphMemory &d_graph) {
  hipMemcpy(host_info.bfs_tree, d_graph.bfs_tree,
      sizeof(int64_t) * host_info.local_v_num, hipMemcpyDeviceToHost);

  for (int64_t v = 0; v < host_info.local_v_num; ++v) { 
    debug_print("v[%ld] 's parent %ld\n", v + host_info.local_v_beg,
        host_info.bfs_tree[v]);
  }
}

void CudaAllocMemory(HostInfo &host_info, CudaGraphMemory &d_graph) {
  // alloc and copy
  hipMalloc((void**)&d_graph.adja_arrays, 
      sizeof(int64_t) * host_info.local_v_num * 2);
  hipMalloc((void**)&d_graph.csr, sizeof(int64_t) * host_info.csr_edge_num);

  hipMemcpy(d_graph.adja_arrays, host_info.adja_arrays,
      sizeof(int64_t) * host_info.local_v_num * 2, hipMemcpyHostToDevice);
  hipMemcpy(d_graph.csr, host_info.csr, 
      sizeof(int64_t) * host_info.csr_edge_num, hipMemcpyHostToDevice);

  // alloc and clear
  hipMalloc((void**)&d_graph.bfs_tree,
      sizeof(int64_t) * host_info.local_v_num);
  hipMalloc((void**)&d_graph.local_bitmap,
      sizeof(bit_type) * host_info.local_v_num);
      /*sizeof(bit_type) * host_info.mpi_local_v_num);*/
  hipMalloc((void**)&d_graph.global_bitmap,
      sizeof(bit_type) * host_info.global_v_num);
      /*sizeof(bit_type) * host_info.mpi_global_v_num);*/
  hipMalloc((void**)&d_graph.p_change, sizeof(bool));
  hipMemset(d_graph.bfs_tree, -1, sizeof(int64_t) * host_info.local_v_num);
  hipMemset(d_graph.local_bitmap, 0, 
      sizeof(bit_type) * host_info.local_v_num);
      /*sizeof(bit_type) * host_info.mpi_local_v_num);*/
  hipMemset(d_graph.global_bitmap, 0,
      sizeof(bit_type) * host_info.global_v_num);
      /*sizeof(bit_type) * host_info.mpi_global_v_num);*/
  hipMemset(d_graph.p_change, 0, sizeof(bool));

  // just copy
  hipMemcpyToSymbol(HIP_SYMBOL(gd_mpi_rank), &host_info.mpi_rank, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_mpi_size), &host_info.mpi_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_num), &host_info.local_v_num, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_global_v_num), &host_info.global_v_num, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_beg), &host_info.local_v_beg, sizeof(int64_t));
  hipMemcpyToSymbol(HIP_SYMBOL(gd_local_v_end), &host_info.local_v_end, sizeof(int64_t));

  int64_t average = host_info.global_v_num / host_info.mpi_size;
  hipMemcpyToSymbol(HIP_SYMBOL(gd_average), &average, sizeof(int64_t));
}

void CudaFreeMemory(CudaGraphMemory &d_graph) {
  hipFree(d_graph.adja_arrays);
  hipFree(d_graph.csr);
  hipFree(d_graph.bfs_tree);
  hipFree(d_graph.local_bitmap);
  hipFree(d_graph.global_bitmap);
  hipFree(d_graph.p_change);
}

__device__ int64_t local_to_global(int64_t local) {
  return gd_local_v_beg + local;
}

__device__ int64_t global_to_local(int64_t global) {
  return global - gd_local_v_beg;
}

__global__ void BFSTopDown() {

}

__global__ void BFSBottomUp(int64_t *adja_arrays,
                            int64_t *csr,
                            int64_t *bfs_tree,
                            bit_type *local_bitmap,
                            bit_type *global_bitmap,
                            bool *p_change) {

  const int64_t local_v_num  = gd_local_v_num;
  const int64_t kThreadsNumber = blockDim.x * gridDim.x;

  *p_change = false;

  for (int64_t local_v = blockIdx.x*blockDim.x + threadIdx.x;
      local_v < local_v_num; local_v += kThreadsNumber) {

    // unvisited
    int64_t global_v = local_to_global(local_v);
    if (-1 == bfs_tree[local_v]) {

      printf("v [%ld] not visited, beg %ld, end %ld\n", 
          global_v, adja_arrays[2*local_v], adja_arrays[2*local_v+1]);

      for (int64_t iter = adja_arrays[2*local_v]; 
          iter < adja_arrays[2*local_v+1]; ++iter) {

        int64_t global_u = csr[iter];
        printf("v[%ld] <-> u[%ld]\n", global_v, global_u);

        // its parent havs been visited
        if (global_bitmap[global_u]) {

          printf("get v[%ld] 's parent global_u %ld\n", global_v, global_u);

          local_bitmap[local_v] = true;
          bfs_tree[local_v] = global_u;

          *p_change = true;
        }
      }
    }
  }
}

void CudaBFS(int mpi_rank,
             int mpi_size,
             int64_t root, 
             int64_t *adja_arrays, 
             int64_t local_v_num, 
             int64_t global_v_num,
             int64_t local_v_beg,
             int64_t local_v_end,
             int64_t *csr,
             int64_t csr_edge_num,
             int64_t *bfs_tree) {

  CudaInfo cuda_info;
  HostInfo host_info = {
    mpi_rank,
    mpi_size,
    root,
    adja_arrays,
    local_v_num,
    global_v_num,
    local_v_beg,
    local_v_end,
    csr,
    csr_edge_num,
    bfs_tree,
    0,
    0,
    0,
    0,
  };
  CudaGraphMemory d_graph;

  InitHostInfo(host_info);

  hipSetDevice(mpi_rank % 4);
  InitCudaDevice(cuda_info);

  HostAllocMemory(host_info);
  CudaAllocMemory(host_info, d_graph);

  /*--------------------------------------------------------------------------*/
  // debug print
  debug_print("mpi_rank: %d\n", host_info.mpi_rank);
  debug_print("mpi_size: %d\n", host_info.mpi_size);
  debug_print("root: %ld\n", host_info.root);
  debug_print("local_v_num: %ld\n", host_info.local_v_num);
  debug_print("global_v_num: %ld\n", host_info.global_v_num);
  debug_print("local_v_beg: %ld\n", host_info.local_v_beg);
  debug_print("local_v_end: %ld\n", host_info.local_v_end);
  debug_print("csr_edge_num: %ld\n", host_info.csr_edge_num);
  debug_print("average: %ld\n", host_info.average);
  debug_print("remainder: %ld\n", host_info.global_v_num % host_info.mpi_size);

  /*--------------------------------------------------------------------------*/

  SetBFSRoot(host_info, d_graph);

  for (int64_t v = 0; v < host_info.local_v_num; ++v) { 
    debug_print("v[%ld] 's parent %ld; global bitmap %d\n", 
        v + host_info.local_v_beg, host_info.bfs_tree[v], 
        host_info.global_bitmap[v+host_info.local_v_beg]);
  }
  debug_print("--------------------------\n");
  MPI_Barrier(MPI_COMM_WORLD);

  for (int64_t u = 0; u < host_info.local_v_num; ++u) {
    int64_t beg = host_info.adja_arrays[2*u],
            end = host_info.adja_arrays[2*u+1];
    debug_print("before cuda u adja beg %ld, end %ld\n", beg, end);
    for (int64_t iter = beg; iter != end; ++iter) {
      debug_print("before cuda: %ld -> %ld\n", 
          u+host_info.local_v_beg, host_info.csr[iter]);
    }
  }

  do {

    if (false) {
      BFSTopDown<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          );
    } else {
      BFSBottomUp<<<cuda_info.blocks_number, cuda_info.threads_per_block>>>(
          d_graph.adja_arrays,
          d_graph.csr,
          d_graph.bfs_tree,
          d_graph.local_bitmap,
          d_graph.global_bitmap,
          d_graph.p_change);
    }

    SyncWithMPI(host_info, d_graph);

  } while (host_info.change);

  CopyBFSTree(host_info, d_graph);

  hipDeviceSynchronize();

  CudaFreeMemory(d_graph);
  HostFreeMemory(host_info);
}


